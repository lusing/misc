#include "hip/hip_runtime.h"
﻿// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

using namespace std;

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(float val) {
    printf("[%d, %d]:\t\tValue is:%f\n", blockIdx.y * gridDim.x + blockIdx.x,
        threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x,
        __sinf(val* threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
            threadIdx.x));
}

__global__ void sqrtKernel(float* in, float* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = sqrtf(in[i]);
        printf("[%d, %d]:\t\tValue is:%f\n", blockIdx.x, threadIdx.x, out[i]);
    }
}

int main(int argc, char** argv) {
    hipDeviceProp_t props;

    int deviceCount;
    hipError_t hipError_t;
    hipError_t = hipGetDeviceCount(&deviceCount);

    if (hipError_t == hipSuccess) {
        cout << "There are " << deviceCount << " cuda devices." << endl;
    }

    for (int i = 0; i < deviceCount; i++)
    {
        hipError_t = hipGetDeviceProperties(&props, i);

        if (hipError_t == hipSuccess) {
            cout << "Device Name： " << props.name << endl;
            cout << "Compute Capability version: " << props.major << "." << props.minor << endl;
            cout << "设备上可用的全局内存总量:(G字节)" << props.totalGlobalMem / 1024 / 1024 / 1024 << endl;
            cout << "时钟频率（以MHz为单位）:" << props.clockRate / 1000 << endl;
            cout << "设备上多处理器的数量:" << props.multiProcessorCount << endl;
            cout << "每个块的最大线程数:" << props.maxThreadsPerBlock <<endl;
            cout << "内存总线宽度(位)" << props.memoryBusWidth << endl;
            cout << "一个块的每个维度的最大尺寸:" << props.maxThreadsDim[0] << ","<< props.maxThreadsDim[1] << "," << props.maxThreadsDim[2] << endl;
            cout << "一个网格的每个维度的最大尺寸:" << props.maxGridSize[0] << "," << props.maxGridSize[1] << "," << props.maxGridSize[2] <<endl;
            //props.block
        }
    }

    // Kernel configuration, where a two-dimensional grid and
    // three-dimensional blocks are configured.
    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 3);
    testKernel << <dimGrid, dimBlock >> > (0.5);
    hipDeviceSynchronize();

    const int n = 1024;
    size_t size = n * sizeof(float);
    float* h_in = (float*)malloc(size);
    float* h_out = (float*)malloc(size);
    float* d_in, * d_out;

    // Initialize input array
    for (int i = 0; i < n; ++i) {
        h_in[i] = (float)i;
    }

    // Allocate device memory
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    // Copy input data to device
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cout << blocksPerGrid << " " << threadsPerBlock << endl;
    sqrtKernel << <blocksPerGrid, threadsPerBlock >> > (d_in, d_out, n);

    // Copy output data to host
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < n; ++i) {
        if (fabsf(h_out[i] - sqrtf(h_in[i])) > 1e-5) {
            printf("Error: h_out[%d] = %f, sqrtf(h_in[%d]) = %f\n", i, h_out[i], i, sqrtf(h_in[i]));
        }
    }

    printf("Success!\n");

    // Free memory
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return EXIT_SUCCESS;
}
