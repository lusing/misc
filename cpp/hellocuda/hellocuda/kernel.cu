#include "hip/hip_runtime.h"
﻿// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

using namespace std;

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(int val) {
    printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
        threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x,
        val);
}

int main(int argc, char** argv) {
    int devID;
    hipDeviceProp_t props;

    int deviceCount;
    hipError_t hipError_t;
    hipError_t = hipGetDeviceCount(&deviceCount);

    if (hipError_t == hipSuccess) {
        cout << "There are " << deviceCount << " cuda devices." << endl;
    }

    for (int i = 0; i < deviceCount; i++)
    {
        hipError_t = hipGetDeviceProperties(&props, i);

        if (hipError_t == hipSuccess) {
            cout << "Device Name： " << props.name << endl;
            cout << "Compute Capability version: " << props.major << "." << props.minor << endl;
            cout << "设备上可用的全局内存总量:(G字节)" << props.totalGlobalMem / 1024 / 1024 / 1024 << endl;
            cout << "时钟频率（以MHz为单位）:" << props.clockRate / 1000 << endl;
            cout << "设备上多处理器的数量:" << props.multiProcessorCount << endl;
            cout << "每个块的最大线程数:" << props.maxThreadsPerBlock <<endl;
            cout << "内存总线宽度(位)" << props.memoryBusWidth << endl;
            cout << "一个块的每个维度的最大尺寸:" << props.maxThreadsDim[0] << ","<< props.maxThreadsDim[1] << "," << props.maxThreadsDim[2] << endl;
            cout << "一个网格的每个维度的最大尺寸:" << props.maxGridSize[0] << "," << props.maxGridSize[1] << "," << props.maxGridSize[2] <<endl;
        }
    }

    // Kernel configuration, where a two-dimensional grid and
    // three-dimensional blocks are configured.
    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 2);
    testKernel << <dimGrid, dimBlock >> > (10);
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
