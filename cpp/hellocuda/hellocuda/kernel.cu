#include "hip/hip_runtime.h"
﻿// System includes
#include <stdio.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

using namespace std;

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(int val) {
    printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
        threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
        threadIdx.x,
        val);
}

int main(int argc, char** argv) {
    int devID;
    hipDeviceProp_t props;

    hipDeviceProp_t deviceProp;
    int deviceCount;
    hipError_t hipError_t;
    hipError_t = hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipError_t = hipGetDeviceProperties(&deviceProp, i);

        cout << "设备 " << i + 1 << " 的主要属性： " << endl;
        cout << "设备显卡型号： " << deviceProp.name << endl;
        cout << "设备全局内存总量（以MB为单位）： " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;
        cout << "设备上一个线程块（Block）中可用的最大共享内存（以KB为单位）： " << deviceProp.sharedMemPerBlock / 1024 << endl;
        cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << deviceProp.regsPerBlock << endl;
        cout << "设备上一个线程块（Block）可包含的最大线程数量： " << deviceProp.maxThreadsPerBlock << endl;
        cout << "设备的计算功能集（Compute Capability）的版本号： " << deviceProp.major << "." << deviceProp.minor << endl;
        cout << "设备上多处理器的数量： " << deviceProp.multiProcessorCount << endl;
    }

    // Get GPU information
    //checkCudaErrors(hipGetDevice(&devID));
    //checkCudaErrors(hipGetDeviceProperties(&props, devID));
    //printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name,
    //    props.major, props.minor);

    //printf("printf() is called. Output:\n\n");

    // Kernel configuration, where a two-dimensional grid and
    // three-dimensional blocks are configured.
    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 2);
    testKernel << <dimGrid, dimBlock >> > (10);
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
